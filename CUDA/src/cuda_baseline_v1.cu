// 1024 thread per block, 1 thread 1 frame

/*
where_is_happiness.wav
        Type  Time(%)      Time     Calls       Avg       Min       Max     Name
GPU activities:   99.94%  4.33811s         1  4.33811s  4.33811s  4.33811s  calculate_pitch

walking.wav
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
GPU activities:   99.93%  9.98657s         1  9.98657s  9.98657s  9.98657s  calculate_pitch
*/


#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdbool.h>

#include <hip/hip_runtime.h>

// WAVE file header format
struct HEADER {
	unsigned char riff[4];						// RIFF string
	unsigned int overall_size	;				// overall size of file in bytes
	unsigned char wave[4];						// WAVE string
	unsigned char fmt_chunk_marker[4];			// fmt string with trailing null char
	unsigned int length_of_fmt;					// length of the format data
	unsigned int format_type;					// format type. 1-PCM, 3- IEEE float, 6 - 8bit A law, 7 - 8bit mu law
	unsigned int channels;						// no.of channels
	unsigned int sample_rate;					// sampling rate (blocks per second)
	unsigned int byterate;						// SampleRate * NumChannels * BitsPerSample/8
	unsigned int block_align;					// NumChannels * BitsPerSample/8
	unsigned int bits_per_sample;				// bits per sample, 8- 8bits, 16- 16 bits etc
	unsigned char data_chunk_header [4];		// DATA string or FLLR string
	unsigned int data_size;						// NumSamples * NumChannels * BitsPerSample/8 - size of the next chunk that will be read
};

#define PI 3.14159265

FILE* ptr;
struct HEADER header;

unsigned char buffer4[4];
unsigned char buffer2[2];
unsigned char buffer1;

int max_freq = 1000;
int min_freq = 40;
const int T = 1024;

void print_wav_header(FILE *ptr);
char* seconds_to_time(float raw_seconds);
long get_num_samples();
long get_size_of_each_sample();
float get_duration_in_seconds();
bool is_valid_sample(long size_of_each_sample);
double frame2volume(int* frame, int frame_size);
int get_median_of_frame(int* frame, int frame_size);

void normalizeAndScale(int audioData[], int length, int desiredBits);
void medianFilter(int* input, int* output, int length, int windowSize);

__global__ void calculate_pitch(int* data, int* pitch, int frame_size, int num_frames, int n1, int n2, int fs) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int start = (bid * T + tid) * frame_size;
    // frame_size: # of samples in a frame
    // num_frames: # of frames
    double acf = 0.0;
    double max_acf = 0.0;
    int max_acf_index = 0;
    for (int shift = n1; shift < n2; shift++) {
        double out = 0.0;
        double deminator = 0.0;
        for (int j = 0; j < frame_size - shift; j++) {
            out += data[start + j] * data[start + j + shift];
            deminator += data[start + j] * data[start + j] + data[start + j + shift] * data[start + j + shift];
        }
        acf = (deminator > 0) ? 2 * out / deminator : 0;
        if (acf > max_acf) {
            max_acf = acf;
            max_acf_index = shift;
        }
    }
    double freq = (double)fs / max_acf_index;
    double semitone = 12 * log2(freq / 440) + 69;
    // if (bid == 100)
        // printf("freq=%f, semitone=%f\n", freq, semitone);
    pitch[bid * T + tid] = round(semitone);
}

int main(int argc, char **argv) {
    int read = 0;

    if (argc < 2) {
        printf("Usage: %s filename.wav\n", argv[0]);
        exit(0);
    }

    ptr = fopen(argv[1], "rb");
    if (ptr == NULL) {
        printf("Error opening file\n");
        exit(1);
    }

    // NOTE: 1. read header parts
    print_wav_header(ptr);

    // NOTE: 2. calculate information about file
    long num_samples = get_num_samples(); // 1: print, 0: don't print
    long size_of_each_sample = get_size_of_each_sample();
    float duration_in_seconds = get_duration_in_seconds();

    if (!is_valid_sample(size_of_each_sample)) {
        exit(1);
    }

    long bytes_in_each_channel = (size_of_each_sample / header.channels);
    char data_buffer[size_of_each_sample];
    long i = 0;
    long low_limit = 0l;
    long high_limit = 0l;
    
    switch (header.bits_per_sample) {
        case 8:
            low_limit = -128;
            high_limit = 127;
            break;
        case 16:
            low_limit = -32768;
            high_limit = 32767;
            break;
        case 32:
            low_limit = -2147483648;
            high_limit = 2147483647;
            break;
    }
    printf("nn.Valid range for data values : %ld to %ld \n", low_limit, high_limit);

    // NOTE: 3. Read data chunks
    int* data[header.channels];
    for (i = 0; i < header.channels; i++) {
        data[i] = (int*)malloc(sizeof(int) * num_samples);
    }
    double avg = 0.0;
    for (i = 0; i < num_samples; i++) {
        read = fread(data_buffer, sizeof(data_buffer), 1, ptr);
        if (read == 1) {
            unsigned int xchannels = 0;
            int data_in_channel = 0;
            int offset = 0; // move the offset for every iteration in the loop below
            for (xchannels = 0; xchannels < header.channels; xchannels++) {
                // NOTE: 4. convert data from little endian to big endian based on bytes in each sample
                if (bytes_in_each_channel == 4) {
                    data_in_channel = data_buffer[offset] | 
                                      (data_buffer[offset + 1] << 8) | 
                                      (data_buffer[offset + 2] << 16) | 
                                      (data_buffer[offset + 3] << 24);
                } else if (bytes_in_each_channel == 2) {
                    data_in_channel = data_buffer[offset] | (data_buffer[offset + 1] << 8);
                } else if (bytes_in_each_channel == 1) {
                    data_in_channel = data_buffer[offset];
                }

                offset += bytes_in_each_channel;
                if (data_in_channel < low_limit || data_in_channel > high_limit) {
                    printf("**value out of range\n");
                    data[xchannels][i] = 0;
                } else {
                    data[xchannels][i] = data_in_channel;
                }
                avg = (avg + (data[xchannels][i] - avg) / (i + 1));
            }
        } else {
            printf("Error reading file. %d bytes\n", read);
            break;
        }
    }
    fclose(ptr);

    // NOTE: 4. Frame blocking
    const int frame_duration = 100; // 100ms
    const int frame_size = header.sample_rate * frame_duration / 1000;
    const int num_frames = num_samples / frame_size;
    printf("frame_size=%d, num_of_frames=%d\n", frame_size, num_frames);
    // NOTE: 5. Calculate volume
    double* volume = (double*)malloc(sizeof(double) * num_frames);
    double volume_threasold = 0.0;
    for (i = 0; i < num_frames; i++) {
        volume[i] = frame2volume(data[0] + i * frame_size, frame_size);
        volume_threasold = fmax(volume_threasold, volume[i]);
        // printf("frame %ld: volume=%f\n", i, volume[i]);
    }
    volume_threasold *= 0.1;
    printf("volume_threasold=%f\n", volume_threasold);
    // NOTE: 6. Calculate max and min frequency for pitch
    int fs = header.sample_rate;
    int n1 = floor(fs / max_freq);
    int n2 = ceil(fs / min_freq);
    printf("n1=%d, n2=%d\n", n1, n2);

    // ADD: Allocate device global memory and copy data to device
    // Input - Only one channel
    int* data_device;
    hipMalloc((void**)&data_device, sizeof(int) * num_samples);
    hipMemcpy(data_device, data[0], sizeof(int) * num_samples, hipMemcpyHostToDevice);
    
    // Output
    int* pitch = (int*)malloc(sizeof(int) * num_frames);
    int* pitch_device;
    hipMalloc((void**)&pitch_device, sizeof(int) * num_frames);

    // ADD: Claim the thread for gpu
    int T = 1024;
    int M = (num_frames + T - 1) / T;
    calculate_pitch<<<M, T>>>(data_device, pitch_device, frame_size, num_frames, n1, n2, fs);

    // ADD: Copy data back to host
    hipMemcpy(pitch, pitch_device, sizeof(int) * num_frames, hipMemcpyDeviceToHost);
    hipFree(data_device);
    hipFree(pitch_device);

    // Smooth the pitch
    int window_size = 10;
    int* pitch_smooth = (int*)malloc(sizeof(int) * num_frames);
    medianFilter(pitch, pitch_smooth, num_frames, window_size);

    for (i=0; i < num_frames; i++) {
        if (volume[i] < volume_threasold) {
            printf("0, ");
        } else {
            printf("%d, ", pitch_smooth[i]);
        }
    }
    // for (i = 0; i < num_frames; i++) {
    //     printf("%d, ", pitch[i]);
    // }

    // free data
    for (i = 0; i < header.channels; i++) {
        free(data[i]);
    }
    free(pitch);
    return 0;
}


void print_wav_header(FILE *ptr) {
	if (ptr == NULL) return;

	int read = 0;

	// read header parts
	// fread: the position of the file pointer is updated automatically after the read operation, 
	// so that successive fread() functions read successive file records.
	read = fread(header.riff, sizeof(header.riff), 1, ptr);
	printf("(1-4): %s \n", header.riff); 

	read = fread(buffer4, sizeof(buffer4), 1, ptr);
	printf("%u %u %u %u\n", buffer4[0], buffer4[1], buffer4[2], buffer4[3]);

	// convert little endian to big endian 4 byte int
	header.overall_size  = buffer4[0] | 
						(buffer4[1]<<8) | 
						(buffer4[2]<<16) | 
						(buffer4[3]<<24);

	printf("(5-8) Overall size: bytes:%u, Kb:%u \n", header.overall_size, header.overall_size/1024);

	read = fread(header.wave, sizeof(header.wave), 1, ptr);
	printf("(9-12) Wave marker: %s\n", header.wave);

	read = fread(header.fmt_chunk_marker, sizeof(header.fmt_chunk_marker), 1, ptr);
	printf("(13-16) Fmt marker: %s\n", header.fmt_chunk_marker);

	read = fread(buffer4, sizeof(buffer4), 1, ptr);
	printf("%u %u %u %u\n", buffer4[0], buffer4[1], buffer4[2], buffer4[3]);

	// convert little endian to big endian 4 byte integer
	header.length_of_fmt = buffer4[0] |
							(buffer4[1] << 8) |
							(buffer4[2] << 16) |
							(buffer4[3] << 24);
	printf("(17-20) Length of Fmt header: %u \n", header.length_of_fmt);

	read = fread(buffer2, sizeof(buffer2), 1, ptr); printf("%u %u \n", buffer2[0], buffer2[1]);

	header.format_type = buffer2[0] | (buffer2[1] << 8);
	char format_name[10] = "";
	if (header.format_type == 1)
	strcpy(format_name,"PCM"); 
	else if (header.format_type == 6)
	strcpy(format_name, "A-law");
	else if (header.format_type == 7)
	strcpy(format_name, "Mu-law");

	printf("(21-22) Format type: %u %s \n", header.format_type, format_name);

	read = fread(buffer2, sizeof(buffer2), 1, ptr);
	printf("%u %u \n", buffer2[0], buffer2[1]);

	header.channels = buffer2[0] | (buffer2[1] << 8);
	printf("(23-24) Channels: %u \n", header.channels);

	read = fread(buffer4, sizeof(buffer4), 1, ptr);
	printf("%u %u %u %u\n", buffer4[0], buffer4[1], buffer4[2], buffer4[3]);

	header.sample_rate = buffer4[0] |
						(buffer4[1] << 8) |
						(buffer4[2] << 16) |
						(buffer4[3] << 24);

	printf("(25-28) Sample rate: %u\n", header.sample_rate);

	read = fread(buffer4, sizeof(buffer4), 1, ptr);
	printf("%u %u %u %u\n", buffer4[0], buffer4[1], buffer4[2], buffer4[3]);

	header.byterate  = buffer4[0] |
						(buffer4[1] << 8) |
						(buffer4[2] << 16) |
						(buffer4[3] << 24);
	printf("(29-32) Byte Rate: %u , Bit Rate:%u\n", header.byterate, header.byterate*8);

	read = fread(buffer2, sizeof(buffer2), 1, ptr);
	printf("%u %u \n", buffer2[0], buffer2[1]);

	header.block_align = buffer2[0] |
					(buffer2[1] << 8);
	printf("(33-34) Block Alignment: %u \n", header.block_align);

	read = fread(buffer2, sizeof(buffer2), 1, ptr);
	printf("%u %u \n", buffer2[0], buffer2[1]);

	header.bits_per_sample = buffer2[0] |
					(buffer2[1] << 8);
	printf("(35-36) Bits per sample: %u \n", header.bits_per_sample);
    
    // ADD: For stupid format of soo and LIST INFO
    read = fread(buffer2, sizeof(buffer2), 1, ptr);
    printf("buffer2: %u %u\n", buffer2[0], buffer2[1]);
    if (buffer2[0] == 100 && buffer2[1] == 97) { // d, a
        // Normal format
        header.data_chunk_header[0] = 'd';
        header.data_chunk_header[1] = 'a';
        read = fread(buffer2, sizeof(buffer2), 1, ptr);
        header.data_chunk_header[2] = 't';
        header.data_chunk_header[3] = 'a';
    } else if (buffer2[0] == 76 && buffer2[1] == 73) { // L, I
        // LIST INFO format
        while (true) {
            read = fread(buffer2, sizeof(buffer2), 1, ptr);
            // printf("buffer2: %u %u\n", buffer2[0], buffer2[1]);
            if (buffer2[0] == 100 && buffer2[1] == 97) { // d, a
                break;
            }
        }
        header.data_chunk_header[0] = 'd';
        header.data_chunk_header[1] = 'a';
        read = fread(buffer2, sizeof(buffer2), 1, ptr);
        header.data_chunk_header[2] = 't';
        header.data_chunk_header[3] = 'a';
    }

	// read = fread(header.data_chunk_header, sizeof(header.data_chunk_header), 1, ptr);
	printf("(37-40) Data Marker: %s \n", header.data_chunk_header);
    printf("data_chunk_header: %u %u %u %u\n", header.data_chunk_header[0], header.data_chunk_header[1], header.data_chunk_header[2], header.data_chunk_header[3]);
	read = fread(buffer4, sizeof(buffer4), 1, ptr);
	printf("%u %u %u %u\n", buffer4[0], buffer4[1], buffer4[2], buffer4[3]);

	header.data_size = buffer4[0] |
				(buffer4[1] << 8) |
				(buffer4[2] << 16) | 
				(buffer4[3] << 24 );
	printf("(41-44) Size of data chunk: %u \n", header.data_size);
}

long get_num_samples() {
	// print data_size, num_channels, bits_per_sample, block_align
	printf("data_size=%u, num_channels=%u, bits_per_sample=%u, block_align=%u \n", header.data_size, header.channels, header.bits_per_sample, header.block_align);
	long num_samples = (8 * header.data_size) / (header.channels * header.bits_per_sample);
	printf("Number of samples:%lu \n", num_samples);
	return num_samples;
}

long get_size_of_each_sample() {
	long size_of_each_sample = (header.channels * header.bits_per_sample) / 8;
	printf("Size of each sample:%ld bytes\n", size_of_each_sample);
	return size_of_each_sample;
}

float get_duration_in_seconds() {
	float duration_in_seconds = (float) header.overall_size / header.byterate;
	printf("Approx.Duration in seconds=%f\n", duration_in_seconds);
	return duration_in_seconds;
}

bool is_valid_sample(long size_of_each_sample) {
	if (header.format_type != 1) return false;
	
	long bytes_in_each_channel = (size_of_each_sample / header.channels);
	// make sure that the bytes-per-sample is completely divisible by num.of channels
	if ((bytes_in_each_channel  * header.channels) != size_of_each_sample) {
		printf("Error: %ld x %ud <> %ld\n", bytes_in_each_channel, header.channels, size_of_each_sample);
		return false;
	}
	return true;
}

char* seconds_to_time(float raw_seconds) {
	char *hms;
	int hours, hours_residue, minutes, seconds, milliseconds;
	hms = (char*) malloc(100);

	sprintf(hms, "%f", raw_seconds);

	hours = (int) raw_seconds/3600;
	hours_residue = (int) raw_seconds % 3600;
	minutes = hours_residue/60;
	seconds = hours_residue % 60;
	milliseconds = 0;

	// get the decimal part of raw_seconds to get milliseconds
	char *pos;
	pos = strchr(hms, '.');
	int ipos = (int) (pos - hms);
	char decimalpart[15];
	memset(decimalpart, ' ', sizeof(decimalpart));
	strncpy(decimalpart, &hms[ipos+1], 3);
	milliseconds = atoi(decimalpart);	


	sprintf(hms, "%d:%d:%d.%d", hours, minutes, seconds, milliseconds);
	return hms;
}

double* note_envelope(double* time_vec) {
    double max_amplitude = 0.9;
    double b = 0.015; // peak_time
    double width = 0.05;
    // double k = 10;
    // double period = 0.05;
    // double exponent = 20;

    // The function = c*t/(t^2+a*t+b^2), with the peak at [b, c/(a+2*b)]
    // Let z=a+4*b, then 50% height occurs at (z-sqrt(z*z-4*b^2))/2 and (z+sqrt(z*z-4*b^2))/2.
    double a = sqrt(width * width + 4 * b * b) - 4 * b; // 0.523
    double c = max_amplitude * (a + 2 * b); // 0.4977
    // get the size of time_vec
    int size = 1000; // TODO: get the size of time_vec
    double* envelope = (double*)malloc(sizeof(double) * size);
    for (int i = 0; i < size; i++) {
        double t = time_vec[i];
        envelope[i] = c * t / (t * t + a * t + b * b); // time=0.022, envelope = 0.089
    }
    return envelope;
}

double frame2volume(int* frame, int frame_size) {
    double volume = 0;
    // abssum
    int median = get_median_of_frame(frame, frame_size);
    // print median
    // printf("median=%f\n", median);
    for (int i = 0; i < frame_size; i++) {
        volume += abs(frame[i] - median);
    }
    return volume;
}

int compare (const void * a, const void * b) {
    return ( *(double*)a - *(double*)b );
}

int int_compare (const void * a, const void * b) {
    return ( *(int*)a - *(int*)b );
}

int get_median_of_frame(int* frame, int frame_size) {
    int* frame_copy = (int*)malloc(sizeof(int) * frame_size);
    memcpy(frame_copy, frame, sizeof(int) * frame_size);
    qsort(frame_copy, frame_size, sizeof(int), compare);
    free(frame_copy);
    return frame_copy[frame_size / 2];
}

void normalizeAndScale(int audioData[], int length, int desiredBits) {
    // Calculate the maximum value for a 16-bit integer
    int max16Bit = (1 << 15);  // Equivalent to (2 ** 16) / 2

    // Calculate the maximum value for the desired bit level
    int maxDesired = (1 << (desiredBits - 1));  // Equivalent to (2 ** desiredBits) / 2

    // Loop through all values, normalize them to 1, then scale to the new max value
    for (int i = 0; i < length; i++) {
        double normalisedSample = (double)audioData[i] / max16Bit;
        int scaledSample = (int)(normalisedSample * maxDesired);
        audioData[i] = scaledSample;
    }
}

void medianFilter(int* input, int* output, int length, int windowSize) {
    int halfWindow = windowSize / 2;
    int windowValues[windowSize];

    for (int i = 0; i < length; i++) {
        // Determine the range of indices for the window
        int start = (i - halfWindow < 0) ? 0 : i - halfWindow;
        int end = (i + halfWindow >= length) ? length - 1 : i + halfWindow;

        // Copy the values within the window to a temporary array
        for (int j = start; j <= end; j++) {
            windowValues[j - start] = input[j];
        }

        // Sort the temporary array (e.g., using bubble sort for simplicity)
        qsort(windowValues, windowSize, sizeof(int), int_compare);

        // Set the output value to the median of the sorted window
        output[i] = windowValues[windowSize / 2];
    }
}

/* GPU properties
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       49152 bytes
  Total number of registers available per block: 65536
  Warp size:                                     32
  Maximum number of threads per multiprocessor:  2048
  Maximum number of threads per block:           1024
  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
*/
